#include "hip/hip_runtime.h"
// 4d integration
// g++ -o integrate test_integrate_feynman.c -lm -lrt
#define TIME_CODE 1
#define CPG 1.6
#define GIG 1000000000
#define PI 3.1415926535897932384626
#define M2 0.0001

#include "gpu_legendreZeros.c"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <omp.h>
#include <time.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/execution_policy.h>


double integrateGaussian4d(int n, double m);
void init_all(double* lZ, double* lWeights, double* lCoeff, double* sinVals, int n, double m);
void cleanUp(double* lZ, double* lWeights, double* lCoeff, double* sinVals, int n);
void sinCalc(double *sinVals, double *vals, int size);

struct permuteFunctor
{
	const double* sinVals_f;
	const double* weights_f;
	double* sumVec_f;
	int n;

	permuteFunctor(thrust::device_vector<double> const& sinV,
				   thrust::device_vector<double> const& gQV,
				   thrust::device_vector<double> & sumV)
	{
		sinVals_f = thrust::raw_pointer_cast(sinV.data());
		weights_f = thrust::raw_pointer_cast(gQV.data());
		sumVec_f = thrust::raw_pointer_cast(sumV.data());
		n = sinV.size();
	}



	__device__
	void operator()(int x)
	{
		int nn = (n*n);
		int nnn = (n*n*n);

		int l = (x % (n));
		int k = (x % (nn)  / (n));
		int j = (x % (nnn) / (nn));
		int i = (x         / (nnn));

		double sumSin = sinVals_f[i] + sinVals_f[j] + sinVals_f[k] + sinVals_f[l] + M2;
		double prodGQ = weights_f[i] * weights_f[j] * weights_f[k] * weights_f[l];

		sumVec_f[(i*(nnn)) + (j*(nn)) + (k*(n)) + l] = (1.0/(sumSin*sumSin)*prodGQ);

	}
};

int main(int argc, char** argv)
{

	int n = 100;
	double m = 0.01;

	if (argc > 1) n = atoi(argv[1]);

	printf("\nResult: %.15f\n", integrateGaussian4d(n,m));

	printf("\n");
	return 0;
}



// Gaussian
double integrateGaussian4d(int n, double m)
{
	// lZ = table of legendreZeros
	// lWeights = table of legendreWeights
	// lCoeff = table of legendre coefficients
	// sinVals = table used to store calculated value of zeros
	double* lZ;
	double* lWeights;
	double* lCoeff;
	double* sinVals;


#if TIME_CODE
	struct timespec time1, time2, elapsed;
	struct timespec diff(struct timespec start, struct timespec end);
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);
#endif


	lZ = new double[n+1];
	lWeights = new double[n+1];
	lCoeff = new double[n+1];
	sinVals = new double[n];

#if TIME_CODE
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);
	elapsed = diff(time1, time2);
	double cputime1 = (double)((double)(CPG)*(double)(GIG * elapsed.tv_sec + elapsed.tv_nsec));
	printf("\n--- Allocation time: %0.6f (msec)\n", cputime1/1000000);
#endif

#if TIME_CODE
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);
#endif
	init_all(lZ, lWeights, lCoeff, sinVals, n, m);
#if TIME_CODE
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);
	elapsed = diff(time1, time2);
	double cputime2 = (double)((double)(CPG)*(double)(GIG * elapsed.tv_sec + elapsed.tv_nsec));
	printf("\n--- Initialization time: %0.6f (msec)\n", cputime2/1000000);
#endif



#if TIME_CODE
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);
#endif
	double sum = 0;
	

/*********************************


				GPU


*********************************/


	thrust::device_vector<double> d_sinVals(n);
	thrust::device_vector<double> d_Weights(n);
	thrust::device_vector<double> d_sumVals(n*n*n*n, 0);

	for (int i = 0; i < n; i ++)
	{
		d_sinVals[i] = sinVals[i];
		d_Weights[i] = lWeights[i];
	}

	thrust::for_each_n(
		thrust::device,
		thrust::counting_iterator<int>(0),
		(n*n*n*n),
		permuteFunctor(d_sinVals, d_Weights, d_sumVals));


	sum = thrust::reduce(d_sumVals.begin(), d_sumVals.end(), (int) 0, thrust::plus<int>());


#if TIME_CODE
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);
	elapsed = diff(time1, time2);
	double cputime3 = (double)((double)(CPG)*(double)(GIG * elapsed.tv_sec + elapsed.tv_nsec));
	printf("\n--- DotProduct time: %0.6f (msec)\n", cputime3/1000000);
#endif


#if TIME_CODE
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);
#endif
	cleanUp(lZ, lCoeff, lWeights, sinVals, n);
#if TIME_CODE
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);
	elapsed = diff(time1, time2);
	double cputime4 = (double)((double)(CPG)*(double)(GIG * elapsed.tv_sec + elapsed.tv_nsec));
	printf("\nClean up time: %0.6f (msec)\n", cputime4/1000000);
#endif

	printf("\n=== Total time: %0.6f (msec)\n", (cputime1+cputime2+cputime3+cputime4)/1000000);

	return sum;
}

void init_all(double* lZ,
			 double* lCoeff,
			 double* lWeights,
			 double* sinVals,
			 int n,
			 double m)
{

// --------------
// Legendre Zeros
// --------------
#if TIME_CODE
	struct timespec time1, time2, elapsed_cpu;
	struct timespec diff(struct timespec start, struct timespec end);
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);
#endif
	getLegendre(lZ, lWeights, lCoeff, n);
#if TIME_CODE
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);
	elapsed_cpu = diff(time1, time2);
	double cputime = (double)((double)(CPG)*(double)(GIG * elapsed_cpu.tv_sec + elapsed_cpu.tv_nsec));
	printf("\n----- Legendre Init time: %0.6f (msec)\n", cputime/1000000);
#endif



// ---------------------------
// Initialize Legendre Weights
// ---------------------------
#if TIME_CODE
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);
#endif
	sinCalc(sinVals,lZ,n);
#if TIME_CODE
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);
	elapsed_cpu = diff(time1, time2);
	cputime = (double)((double)(CPG)*(double)(GIG * elapsed_cpu.tv_sec + elapsed_cpu.tv_nsec));
	printf("\n----- Weight Init time: %0.6f (msec)\n", cputime/1000000);
#endif
}



void sinCalc(double *sinVals, double *vals, int size)
{
	for(int i = 0 ; i < size; i++)
	{
		sinVals[i] = sin(PI * vals[i]/2) * sin(PI * vals[i]/2);
	}
}

void cleanUp(double* lZ,
			 double* lWeights,
			 double* lCoeff,
			 double* sinVals,
			 int n)
{
	int i;
	delete[] lZ;
	delete[] lWeights;
	delete[] lCoeff;
	delete[] sinVals;
	return;
}

struct timespec diff(struct timespec start, struct timespec end)
{
  struct timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return temp;
}